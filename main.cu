#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>
using namespace cv;

__global__
void colorToGrayScale(int height, int width, int channels, unsigned char* d_img_in, unsigned char* d_img_out) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int offset = row * width + col;

        int rbg_offset = offset * channels;

        unsigned char r = d_img_in[rbg_offset    ];
        unsigned char g = d_img_in[rbg_offset + 1];
        unsigned char b = d_img_in[rbg_offset + 2];

        d_img_out[offset] = r * 0.21f + g * 0.71f + b * 0.07f;
    }
}

int main() {
    std::string image_path = samples::findFile("teste.jpg");
    Mat img = imread(image_path, IMREAD_COLOR);

    if (img.empty()) {
        std::cout << "Could not read image" << std::endl;
        return 1;
    }

    int height = img.size().height;
    int width = img.size().width;
    int channels = img.channels();

    std::cout << "channels: " << channels << " width: " << width << " height: " << height << std::endl;

    cvtColor(img, img, COLOR_BGR2RGB, 0);

    int rows, cols;
    if (img.isContinuous()) {
        rows = 1;
        cols = height * width * channels;
    } else {
        rows = height;
        cols = width;
    }

    std::cout << "building img_vals" << std::endl;
    int cuda_img_size = height * width * channels;
    std::cout << "cuda_size: " << cuda_img_size << std::endl;
    unsigned char* img_vals = new unsigned char[cuda_img_size];
    uchar* p;
    std::cout << "hi" << std::endl;
    for (int row = 0; row < rows; row++) {
        p = img.ptr<uchar>(row);
        for (int col = 0; col < cols; col++) {
            int offset = row * width + col;
            img_vals[col] = p[col];
        }
    }

    std::cout << "allocating cuda stuff" << std::endl;
    unsigned char *d_img_in; hipMallocManaged(&d_img_in, cuda_img_size * sizeof(unsigned char));
    unsigned char *d_img_out; hipMallocManaged(&d_img_out, height * width * sizeof(unsigned char));

    std::cout << "doing copy stuff into cuda stuff" << std::endl;
    std::copy(&img_vals[0], &img_vals[0] + cuda_img_size, d_img_in);

    std::cout << "kernal funsies" << std::endl;
    dim3 dimGrid(ceil (height/16.0), ceil(width/16.0), 1);
    dim3 dimBlock(16, 16, 1);
    colorToGrayScale<<<dimGrid, dimBlock>>>(height, width, channels, (unsigned char *)d_img_in, (unsigned char *)d_img_out);

    hipDeviceSynchronize();

    std::cout << "building out image" << std::endl;
    Mat out_img(height, width, CV_8UC1, Scalar(0, 0, 0));

    rows = 1;
    cols = height * width;
    for (int row = 0; row < rows; row++) {
        p = out_img.ptr<uchar>(row);
        for (int col = 0; col < cols; col++) {
            int offset = row * width + col;
            p[col] = d_img_out[col];
        }
    }

    imwrite("cuda_dump/test2.jpg", out_img);


    std::cout << "saved image" << std::endl;
    return 0;
}
